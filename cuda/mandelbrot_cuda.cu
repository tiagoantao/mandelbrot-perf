#include "hip/hip_runtime.h"
#include "shared.h"

#include <stdio.h>
#include <stdlib.h>

#define MAX_ITER 1000


__global__
void compute_mandelbrot_vertical(int width, int height, uint8_t* area) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    double cx = ((double)x / width) * 4.0 - 2.0;
    for (int y=0; y<height; y++)  {
        double cy = ((double)y / height) * 4.0 - 2.0;
        double zx, zy, zx2, zy2;
        zx = 0.0;
        zy = 0.0;
        zx2 = zx * zx;
        zy2 = zy * zy;
        int iter = 0;
        while (iter < MAX_ITER && zx2 + zy2 < 4.0) {
            zy = 2.0 * zx * zy + cy;
            zx = zx2 - zy2 + cx;
            zx2 = zx * zx;
            zy2 = zy * zy;
            iter++;
        }
        area[y * width + x] = iter;
    }
}


uint8_t* compute_all_mandelbrot(int width, int height) {
    uint8_t* garea;
    uint8_t* area;
    int size = width * height * sizeof(uint8_t);
    area = (uint8_t*)malloc(size);
    hipMallocManaged(&garea, size);
    compute_mandelbrot_vertical<<<width/256, 256>>>(width, height, garea);
    hipDeviceSynchronize();
    hipMemcpy(area, garea, size, hipMemcpyDeviceToHost);
    hipFree(garea);
    return area;
}


int main(int argc, char *argv[]) {
    uint8_t* mandel_area;
    ScreenSize screen_size = get_screen_size(argc, argv);

    printf("Width: %d, Height: %d\n", screen_size.width, screen_size.height);
    mandel_area = compute_all_mandelbrot(screen_size.width, screen_size.height);

    write_mandel_image("cuda.png", screen_size, mandel_area);

    free(mandel_area);

    return 0;
}
